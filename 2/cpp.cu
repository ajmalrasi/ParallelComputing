#include<algorithm>
#include<iostream>
#include<vector>
#include<stdio.h>

#include "hip/hip_runtime.h"


__global__ void add_atomic(int *g)
{
	// which thread is this?
	int i = blockIdx.x * blockDim.x + threadIdx.x; 

	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
	// i = i % ARRAY_SIZE;  
	// atomicAdd(&g[i], 1);
	// i = i % 80;  

	g[i] = threadIdx.x;
}

int main() {
    const int SIZE = 16;
    int *arr = new int[SIZE];
    int *arr_d, *inp_d;
    std::fill_n(arr, SIZE, 10);

    for (size_t i = 0; i < SIZE; i++) {
       std::cout << arr[i] << std::endl;
    }
    
    // std::cout << sizeof(arr) * SIZE <<std::endl;
    hipMalloc((void **) &arr_d, sizeof(arr) * SIZE);
    // cudaMemset((void *)arr_d, 0, sizeof(arr) * SIZE);
    std::cout << "============" <<std::endl;

    hipMemcpy(arr_d, arr, sizeof(arr) * SIZE, hipMemcpyHostToDevice);
    add_atomic<<<3, 1,SIZE>>>(arr_d);
    hipMemcpy(arr, arr_d, sizeof(arr) * SIZE, hipMemcpyDeviceToHost);

    for (size_t i = 0; i < SIZE; i++) {
       std::cout << arr[i] << std::endl;
    }
    return 0;
}