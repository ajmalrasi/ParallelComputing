#include <stdio.h>
#include "time.h"
#include <unistd.h>

#include "hip/hip_runtime.h"


#define NUM_THREADS 1000000
#define ARRAY_SIZE  100

#define BLOCK_WIDTH 1000

void print_array(int *array, int size)
{
    printf("{ ");
    for (int i = 0; i < size; i++)  { printf("%d ", array[i]); }
    printf("}\n");
}

__global__ void increment_naive(int *g)
{
	// which thread is this?
	int i = blockIdx.x * blockDim.x + threadIdx.x; 

	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
	i = i % ARRAY_SIZE;  
	g[i] = g[i] + 1;
}

__global__ void increment_atomic(int *g)
{
	// which thread is this?
	int i = blockIdx.x * blockDim.x + threadIdx.x; 

	// each thread to increment consecutive elements, wrapping at ARRAY_SIZE
	i = i % ARRAY_SIZE;  
	atomicAdd(& g[i], 1);
}

int main(int argc,char **argv)
{   
    clock_t begin, end;
    printf("%d total threads in %d blocks writing into %d array elements\n",
           NUM_THREADS, NUM_THREADS / BLOCK_WIDTH, ARRAY_SIZE);

    // declare and allocate host memory
    int h_array[ARRAY_SIZE];
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);
 
    // declare, allocate, and zero out GPU memory
    int *d_array;
    hipMalloc((void **) &d_array, ARRAY_BYTES);
    hipMemset((void *) d_array, 0, ARRAY_BYTES); 

    // launch the kernel - comment out one of these
    begin =  clock();
    // sleep(10);
    
    
    // Instructions: This program is needed for the next quiz
    // uncomment increment_naive to measure speed and accuracy 
    // of non-atomic increments or uncomment increment_atomic to
    // measure speed and accuracy of  atomic icrements
    // increment_naive<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_array);
    increment_atomic<<<NUM_THREADS/BLOCK_WIDTH, BLOCK_WIDTH>>>(d_array);
    end =  clock();
    
    // copy back the array of sums from GPU and print
    hipMemcpy(h_array, d_array, ARRAY_BYTES, hipMemcpyDeviceToHost);
    print_array(h_array, ARRAY_SIZE);
    printf("Time elapsed = %f ms\n", (double)(end - begin)/8);
 
    // free GPU memory allocation and exit
    hipFree(d_array);
    return 0;
}